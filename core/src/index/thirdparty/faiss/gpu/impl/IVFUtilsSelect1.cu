#include "hip/hip_runtime.h"
/**
 * Copyright (c) Facebook, Inc. and its affiliates.
 *
 * This source code is licensed under the MIT license found in the
 * LICENSE file in the root directory of this source tree.
 */


#include <faiss/gpu/impl/IVFUtils.cuh>
#include <faiss/gpu/utils/DeviceDefs.cuh>
#include <faiss/gpu/utils/DeviceUtils.h>
#include <faiss/gpu/utils/Limits.cuh>
#include <faiss/gpu/utils/Select.cuh>
#include <faiss/gpu/utils/StaticUtils.h>
#include <faiss/gpu/utils/Tensor.cuh>

//
// This kernel is split into a separate compilation unit to cut down
// on compile time
//

namespace faiss { namespace gpu {

template <int ThreadsPerBlock, int NumWarpQ, int NumThreadQ, bool Dir>
__global__ void
pass1SelectLists(Tensor<int, 2, true> prefixSumOffsets,
                 Tensor<uint8_t, 1, true> bitset,
                 Tensor<float, 1, true> distance,
                 int nprobe,
                 int k,
                 Tensor<float, 3, true> heapDistances,
                 Tensor<int, 3, true> heapIndices) {
  constexpr int kNumWarps = ThreadsPerBlock / kWarpSize;

  __shared__ float smemK[kNumWarps * NumWarpQ];
  __shared__ int smemV[kNumWarps * NumWarpQ];

  constexpr auto kInit = Dir ? kFloatMin : kFloatMax;
  BlockSelect<float, int, Dir, Comparator<float>,
              NumWarpQ, NumThreadQ, ThreadsPerBlock>
    heap(kInit, -1, smemK, smemV, k);

  auto queryId = blockIdx.y;
  auto sliceId = blockIdx.x;
  auto numSlices = gridDim.x;

  int sliceSize = (nprobe / numSlices);
  int sliceStart = sliceSize * sliceId;
  int sliceEnd = sliceId == (numSlices - 1) ? nprobe :
    sliceStart + sliceSize;
  auto offsets = prefixSumOffsets[queryId].data();

  // We ensure that before the array (at offset -1), there is a 0 value
  int start = *(&offsets[sliceStart] - 1);
  int end = offsets[sliceEnd - 1];

  int num = end - start;
  int limit = utils::roundDown(num, kWarpSize);

  int i = threadIdx.x;
  auto distanceStart = distance[start].data();
  bool bitsetEmpty = (bitset.getSize(0) == 0);
  int blockId, threadId;
  int idx;

  // BlockSelect add cannot be used in a warp divergent circumstance; we
  // handle the remainder warp below
  for (; i < limit; i += blockDim.x) {
    /* 2D-grid, 1D-block */
    //blockId = blockIdx.x + blockIdx.y * gridDim.x + blockIdx.z * gridDim.x * gridDim.y;
    //threadId = blockId * (blockDim.x * blockDim.y * blockDim.z) + (threadIdx.z * (blockDim.x * blockDim.y))
    //           + (threadIdx.y * blockDim.x) + threadIdx.x;
    blockId = blockIdx.y * gridDim.x + blockIdx.x;
    threadId = blockId * blockDim.x + i;
    idx = start + i;
//      printf("CYD(%s:%d) - gridDim(%d, %d, %d), blockDim(%d, %d, %d), blockIdx(%d, %d, %d), threadIdx(%d, %d),"
//             "start = %d, i = %d, blockId = %d, threadId = %d, bitset %s\n",
//             __FUNCTION__, __LINE__,
//             gridDim.x, gridDim.y, gridDim.z,
//             blockDim.x, blockDim.y, blockDim.z,
//             blockIdx.x, blockIdx.y, blockIdx.z,
//             threadIdx.x, threadIdx.y,
//             start, i, blockId, threadId, ((bitset[threadId >> 3] & (0x1 << (threadId & 0x7))))? "TRUE ===========" : "FALSE");
    if (bitsetEmpty || (!(bitset[threadId >> 3] & (0x1 << (threadId & 0x7))))) {
      heap.add(distanceStart[i], start + i);
//      printf("CYD - add %f %d\n", distanceStart[i], start + i);
    } else {
      heap.add((1.0 / 0.0), start + i);
//      printf("CYD - add %f %d\n", (1.0/0.0), start + i);
    }
  }

  // Handle warp divergence separately
  if (i < num) {
    blockId = blockIdx.y * gridDim.x + blockIdx.x;
    threadId = blockId * blockDim.x + i;
    if (bitsetEmpty || (!(bitset[threadId >> 3] & (0x1 << (threadId & 0x7))))) {
      heap.addThreadQ(distanceStart[i], start + i);
    } else {
      heap.addThreadQ((1.0 / 0.0), start + i);
//      printf("CYD(%s:%d) - blockDim(%d, %d), blockIdx(%d, %d), threadIdx(%d, %d), i = %d, blockId = %d, threadId = %d, bitset %s\n",
//             __FUNCTION__, __LINE__, blockDim.x, blockDim.y, blockIdx.x, blockIdx.y, threadIdx.x, threadIdx.y,
//             i, blockId, threadId, ((bitset[threadId >> 3] & (0x1 << (threadId & 0x7))))? "TRUE ===========" : "FALSE");
    }
  }

  // Merge all final results
  heap.reduce();

  // Write out the final k-selected values; they should be all
  // together
  for (int i = threadIdx.x; i < k; i += blockDim.x) {
    printf("XY - i = %d, %d\n", i, smemV[i]);
    heapDistances[queryId][sliceId][i] = smemK[i];
    heapIndices[queryId][sliceId][i] = smemV[i];
  }
}

void
runPass1SelectLists(Tensor<int, 2, true>& prefixSumOffsets,
                    Tensor<uint8_t, 1, true>& bitset,
                    Tensor<float, 1, true>& distance,
                    int nprobe,
                    int k,
                    bool chooseLargest,
                    Tensor<float, 3, true>& heapDistances,
                    Tensor<int, 3, true>& heapIndices,
                    hipStream_t stream) {
  // This is caught at a higher level
  FAISS_ASSERT(k <= GPU_MAX_SELECTION_K);

  auto grid = dim3(heapDistances.getSize(1), prefixSumOffsets.getSize(0));

#define RUN_PASS(BLOCK, NUM_WARP_Q, NUM_THREAD_Q, DIR)                  \
  do {                                                                  \
    pass1SelectLists<BLOCK, NUM_WARP_Q, NUM_THREAD_Q, DIR>              \
      <<<grid, BLOCK, 0, stream>>>(prefixSumOffsets,                    \
                                   bitset,                              \
                                   distance,                            \
                                   nprobe,                              \
                                   k,                                   \
                                   heapDistances,                       \
                                   heapIndices);                        \
    CUDA_TEST_ERROR();                                                  \
    return; /* success */                                               \
  } while (0)

#if GPU_MAX_SELECTION_K >= 2048

  // block size 128 for k <= 1024, 64 for k = 2048
#define RUN_PASS_DIR(DIR)                                 \
  do {                                                    \
    if (k == 1) {                                         \
      RUN_PASS(128, 1, 1, DIR);                           \
    } else if (k <= 32) {                                 \
      RUN_PASS(128, 32, 2, DIR);                          \
    } else if (k <= 64) {                                 \
      RUN_PASS(128, 64, 3, DIR);                          \
    } else if (k <= 128) {                                \
      RUN_PASS(128, 128, 3, DIR);                         \
    } else if (k <= 256) {                                \
      RUN_PASS(128, 256, 4, DIR);                         \
    } else if (k <= 512) {                                \
      RUN_PASS(128, 512, 8, DIR);                         \
    } else if (k <= 1024) {                               \
      RUN_PASS(128, 1024, 8, DIR);                        \
    } else if (k <= 2048) {                               \
      RUN_PASS(64, 2048, 8, DIR);                         \
    }                                                     \
  } while (0)

#else

#define RUN_PASS_DIR(DIR)                                 \
  do {                                                    \
    if (k == 1) {                                         \
      RUN_PASS(128, 1, 1, DIR);                           \
    } else if (k <= 32) {                                 \
      RUN_PASS(128, 32, 2, DIR);                          \
    } else if (k <= 64) {                                 \
      RUN_PASS(128, 64, 3, DIR);                          \
    } else if (k <= 128) {                                \
      RUN_PASS(128, 128, 3, DIR);                         \
    } else if (k <= 256) {                                \
      RUN_PASS(128, 256, 4, DIR);                         \
    } else if (k <= 512) {                                \
      RUN_PASS(128, 512, 8, DIR);                         \
    } else if (k <= 1024) {                               \
      RUN_PASS(128, 1024, 8, DIR);                        \
    }                                                     \
  } while (0)

#endif // GPU_MAX_SELECTION_K

  if (chooseLargest) {
    RUN_PASS_DIR(true);
  } else {
    RUN_PASS_DIR(false);
  }

#undef RUN_PASS_DIR
#undef RUN_PASS
}

} } // namespace
